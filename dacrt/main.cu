#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_vector_types.h>
#include <algorithm>

const int HIT_EPSILON = 1e-6f;

struct triangle
{
	float3 v0;
	float3 v1;
	float3 v2;
};

__global__ void kernel(triangle* tris, float3 rsrc, float3* rdirs, int* hits, int numTiles)
{
	extern __shared__ triangle sh_tris[];

	float bestDist = 3.40282347E+38F;
	int bestTri = -1;

	int gtid = blockIdx.x*blockDim.x+threadIdx.x;
	float3 rdir = rdirs[gtid];

	for(int tile = 0; tile < numTiles; ++tile)
	{
		// load triangles to shared memory
		sh_tris[threadIdx.x] = tris[tile*blockDim.x+threadIdx.x];

		__syncthreads();

		// compute intersections
		for(unsigned int tid = 0; tid < blockDim.x; ++tid)
		{
			triangle tri = sh_tris[tid];

			/* find vectors for two edges sharing vert0 */
			const float3 edge1 = tri.v1 - tri.v0;
			const float3 edge2 = tri.v2 - tri.v0;

			/* begin calculating determinant - also used to calculate U parameter */
			const float3 pvec = cross(rdir, edge2);

			/* if determinant is near zero, ray lies in plane of triangle */
			float det = dot(edge1, pvec);

			if(det > -HIT_EPSILON && det < HIT_EPSILON)
				continue;

			const float inv_det = 1.0f / det;

			/* calculate distance from vert0 to ray origin */
			const float3 tvec = rsrc - tri.v0;

			/* calculate U parameter and test bounds */
			const float u = dot(tvec, pvec) * inv_det;
			if(u < 0.0f || u > 1.0f)
				continue;

			/* prepare to test V parameter */
			const float3 qvec = cross(tvec, edge1);

			/* calculate V parameter and test bounds */
			const float v = dot(rdir, qvec) * inv_det;
			if(v < 0.0f || u + v > 1.0f)
				continue;

			/* calculate t, ray hits triangle */
			const float f = dot(edge2, qvec) * inv_det;

			if((f >= bestDist) || (f < -HIT_EPSILON))
				continue;

			// Have a valid hit point here. Store it.
			bestDist = f;
			bestTri = tid;
		}

		__syncthreads();
	}

	// write final result
	hits[gtid] = bestTri;
}

float randf(float min, float max)
{
	return (float)rand()/(float)RAND_MAX * (max - min);
}

int main()
{
	const int ntri = 4096;
	const int nray = 4096;
    const int repeat = 5;

    float time, cumulative_time = 0.f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    srand(122);

    triangle* h_tris;
    hipHostMalloc(&h_tris, ntri*sizeof(triangle));
    triangle* d_tris;
    hipMalloc(&d_tris, ntri*sizeof(triangle));
    for(int i = 0; i < ntri; ++i)
    {
    	float3 c = make_float3(randf(-5, 5),randf(-5, 5), randf(-5, 5));
    	triangle t;
    	t.v0 = c + make_float3(randf(0.1f, 0.5f), randf(0.1f, 0.5f), randf(0.1f, 0.5f));
    	t.v1 = c + make_float3(randf(0.1f, 0.5f), randf(0.1f, 0.5f), randf(0.1f, 0.5f));
    	t.v2 = c + make_float3(randf(0.1f, 0.5f), randf(0.1f, 0.5f), randf(0.1f, 0.5f));
    	h_tris[i] = t;
    }
    hipMemcpy(d_tris, h_tris, ntri*sizeof(triangle), hipMemcpyDefault);

    float3 src = make_float3(0,0,10);

    float3* h_dirs;
    hipHostMalloc(&h_dirs, nray*sizeof(float3));
    float3* d_dirs;
    hipMalloc(&d_dirs, nray*sizeof(float3));
    for(int i = 0; i < nray; ++i)
    {
    	float3 dir = make_float3(randf(-10, 10), randf(-10, 10), 0.0);
    	h_dirs[i] = dir - src;
    }
    hipMemcpy(d_dirs, h_dirs, nray*sizeof(float3), hipMemcpyDefault);

    int* h_hits;
    hipHostMalloc(&h_hits, nray*sizeof(int));
    int* d_hits;
    hipMalloc(&d_hits, nray*sizeof(int));
    std::fill_n(h_hits, nray, -1);
    hipMemcpy(d_hits, h_hits, nray*sizeof(int), hipMemcpyDefault);

    int blockSize = 1024; // 1024
    int numBlocks = nray / blockSize;
    int numTiles = ntri / blockSize;
    int sharedMemSize = blockSize * sizeof(triangle);

    for(int i=0; i<repeat; i++)
    {
        hipEventRecord(start, 0);
        kernel<<<numBlocks, blockSize, sharedMemSize>>>(d_tris, src, d_dirs, d_hits, numTiles);
        hipEventRecord(stop, 0);

        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        cumulative_time = cumulative_time + time;
    }

    printf("Kernel time:  %3.5f ms \n", cumulative_time / repeat);

    hipMemcpy(h_hits, d_hits, nray*sizeof(int), hipMemcpyDefault);
    int nhits = 0;
    for(int i = 0; i < nray; ++i)
    {
    	if(h_hits[i] >= 0)
    	{
    		++nhits;
    	}
    }

    printf("nhits: %d\n", nhits);

    return 0;
}
