#include "hip/hip_runtime.h"
#include "rtv.h"
#include "rply.h"
#include "timer.h"
#include <cfloat>
#include <algorithm>
#include <iostream>
#include <iomanip>
#include <vector>
#include <fstream>

//#define HIT_CPU
#define HIT_GPU

#define HIT_EPSILON 1e-6f
#define BLOCKSIZE 256

// ---------------------------------------------------------------------------------------------------------------------------------------------------------------------------

__host__ __device__ inline float minf(float a, float b)
{
	return (a < b)? a : b;
}

__host__ __device__ inline float maxf(float a, float b)
{
	return (a > b)? a : b;
}

struct vec3
{
	float x,y,z;
	__host__ __device__ vec3() {}
	__host__ __device__ explicit vec3(float a) : x(a), y(a), z(a) {}
	__host__ __device__ explicit vec3(const float* const p) : x(p[0]), y(p[1]), z(p[2]) {}
	__host__ __device__ vec3(float a, float b, float c) : x(a), y(b), z(c) {}
	__host__ __device__ float* ptr() { return &x; }
	__host__ __device__ const float* ptr() const { return &x; }
	__host__ __device__ float& operator[](int i) { return (&x)[i]; }
	__host__ __device__ float operator[](int i) const { return (&x)[i]; }
	__host__ __device__ vec3 operator-() const { return vec3(-x, -y, -z); }
	__host__ __device__ vec3 operator+(const vec3& o) const { return vec3(x+o.x, y+o.y, z+o.z); }
	__host__ __device__ vec3 operator-(const vec3& o) const { return vec3(x-o.x, y-o.y, z-o.z); }
	__host__ __device__ vec3 operator*(float a) const { return vec3(x*a, y*a, z*a); }
	__host__ __device__ void operator+=(const vec3& o) { x+=o.x; y+=o.y; z+=o.z; }
	__host__ __device__ void operator*=(float a) { x*=a; y*=a; z*=a; }
	__host__ __device__ void setmin(const vec3& o) { x = minf(x, o.x); y = minf(y, o.y); z = minf(z, o.z); }
	__host__ __device__ void setmax(const vec3& o) { x = maxf(x, o.x); y = maxf(y, o.y); z = maxf(z, o.z); }
};
__host__ __device__ inline float dot(const vec3& a, const vec3& b) { return a.x*b.x + a.y*b.y + a.z*b.z; }
__host__ __device__ inline vec3 cross(const vec3& a, const vec3& b) { return vec3(a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x); }
__host__ __device__ inline float len(const vec3& v) { return sqrtf(v.x*v.x + v.y*v.y + v.z*v.z); }
__host__ __device__ inline vec3 normalize(const vec3& v) { const float invlen = 1.0f / len(v); return vec3(v.x*invlen, v.y*invlen, v.z*invlen); }
__host__ __device__ inline vec3 reciprocal(const vec3& v) { return vec3(1.0f/v.x, 1.0f/v.y, 1.0f/v.z); }

// ---------------------------------------------------------------------------------------------------------------------------------------------------------------------------

struct TriV // 36 bytes
{
	vec3 v[3];
};

struct TriN // 36 bytes
{
	vec3 n[3];
};

struct Ray // 28 bytes
{
	vec3 o;
	vec3 d;
	int id;
};

struct Box // 24 bytes
{
	vec3 min, max;
	Box() : min(FLT_MAX), max(-FLT_MAX) {}
	void expand(const vec3& v)
	{
		min.setmin(v);
		max.setmax(v);
	}
	void expand(const TriV& tri)
	{
		expand(tri.v[0]);
		expand(tri.v[1]);
		expand(tri.v[2]);
	}
};

struct TriSet
{
	Box boxTris;
	Box boxCenters;
	int count;
	int* ids;
	Box* boxes;
	vec3* centers;
	TriV* verts;
	TriN* norms;
};

struct RaySet
{
	int count;
	int* ids;
	Ray* rays;
	float* tmaxs;
	int* hits;
};

// ---------------------------------------------------------------------------------------------------------------------------------------------------------------------------

static timer s_t;
static double s_intersectTime = 0.0;
static double s_triSendTime = 0.0;
static double s_raySendTime = 0.0;
static double s_kernelTime = 0.0;
static double s_splitTime = 0.0;
static double s_partitionTriTime = 0.0;
static double s_filterRayTime = 0.0;
#define TIMED_CALL(f, t) s_t.restart(); f; t += s_t.msec();
static int s_numIntersectCalls = 0;

// ---------------------------------------------------------------------------------------------------------------------------------------------------------------------------

static TriSet g_tris;
static RaySet g_rays;

static TriV* d_tris = 0;
static Ray* d_rays = 0;
static float* d_tmaxs = 0;
static int* d_hits = 0;

static TriV* h_tris = 0;
static Ray* h_rays = 0;

// ---------------------------------------------------------------------------------------------------------------------------------------------------------------------------

__global__ void kernel(TriV* tris, int tbegin, int numTiles, int lastTileSize, Ray* rays, int numRays, float* tmaxs, int* hits)
{
	extern __shared__ TriV sh_tris[];

	int idx = blockIdx.x*blockDim.x+threadIdx.x;

	Ray ray;
	float tmax;
	if(idx < numRays)
	{
		ray = rays[idx];
		tmax = tmaxs[ray.id];
	}

	int hit = -1;

	for(int tile = 0; tile < numTiles; ++tile)
	{
		const bool isLastTile = (tile == numTiles-1);

		if(!isLastTile || threadIdx.x < lastTileSize)
		{
			sh_tris[threadIdx.x] = tris[tile*blockDim.x+threadIdx.x];
		}

		__syncthreads();

		if(idx < numRays)
		{
			const int limit = isLastTile ? lastTileSize : blockDim.x;
			for(int t = 0; t < limit; ++t)
			{
				TriV triv = sh_tris[t];

				// find vectors for two edges sharing vert0
				const vec3 edge1 = triv.v[1] - triv.v[0];
				const vec3 edge2 = triv.v[2] - triv.v[0];

				// begin calculating determinant - also used to calculate U parameter
				const vec3 pvec = cross(ray.d, edge2);

				// if determinant is near zero, ray lies in plane of triangle
				const float det = dot(edge1, pvec);

				if(det > -HIT_EPSILON && det < HIT_EPSILON)
					continue;

				const float inv_det = 1.0f / det;

				// calculate distance from vert0 to ray origin
				const vec3 tvec = ray.o - triv.v[0];

				// calculate U parameter and test bounds
				const float u = dot(tvec, pvec) * inv_det;
				if(u < 0.0f || u > 1.0f)
					continue;

				// prepare to test V parameter
				const vec3 qvec = cross(tvec, edge1);

				// calculate V parameter and test bounds
				const float v = dot(ray.d, qvec) * inv_det;
				if(v < 0.0f || u + v > 1.0f)
					continue;

				// calculate t, ray hits triangle
				const float f = dot(edge2, qvec) * inv_det;

				if(f >= tmax || f < -HIT_EPSILON)
					continue;

				// store valid hit
				tmax = f;
				hit = tile*blockDim.x + t;
			}
		}

		__syncthreads();
	}

	if(idx < numRays)
	{
		tmaxs[ray.id] = tmax;
		if(hit >= 0)
			hits[ray.id] = hit + tbegin;
	}
}

void intersectGPU(int tbegin, int tend, int rend)
{
	static hipStream_t stream;
	static hipEvent_t copyEvent;
	static hipEvent_t kernelEvent;
	static bool first = true;
	if(first)
	{
		hipStreamCreate(&stream);
		hipEventCreate(&copyEvent);
		hipEventCreate(&kernelEvent);
		first = false;
	}

	const int tcount = tend - tbegin;
	const int numBlocks = ceilf((float)rend/(float)BLOCKSIZE);
	const int numTiles = ceilf((float)tcount/(float)BLOCKSIZE);
	const int lastTileSize = tcount < BLOCKSIZE? tcount : tcount % BLOCKSIZE;

	hipEventSynchronize(copyEvent);

	std::copy(g_tris.verts + tbegin, g_tris.verts + tend, h_tris);
	std::copy(g_rays.rays, g_rays.rays + rend, h_rays);

	hipEventSynchronize(kernelEvent);

	hipMemcpy(g_rays.tmaxs, d_tmaxs, g_rays.count*sizeof(float), hipMemcpyDefault);

	TIMED_CALL(hipMemcpyAsync(d_tris, h_tris, tcount*sizeof(TriV), hipMemcpyDefault, stream), s_triSendTime);
	TIMED_CALL(hipMemcpyAsync(d_rays, h_rays, rend*sizeof(Ray), hipMemcpyDefault, stream), s_raySendTime);

	hipEventRecord(copyEvent, stream);

	s_t.restart();
	kernel<<<numBlocks, BLOCKSIZE, BLOCKSIZE*sizeof(TriV), stream>>>(d_tris, tbegin, numTiles, lastTileSize, d_rays, rend, d_tmaxs, d_hits);
	s_kernelTime += s_t.msec();

	hipEventRecord(kernelEvent, stream);

	++s_numIntersectCalls;
}

void intersectCPU(int tbegin, int tend, int rend)
{
	for(int t = tbegin; t < tend; ++t)
	{
		const TriV& triv = g_tris.verts[t];

		for(int r = 0; r < rend; ++r)
		{
			Ray& ray = g_rays.rays[r];

			// find vectors for two edges sharing vert0
			const vec3 edge1 = triv.v[1] - triv.v[0];
			const vec3 edge2 = triv.v[2] - triv.v[0];

			// begin calculating determinant - also used to calculate U parameter
			const vec3 pvec = cross(ray.d, edge2);

			// if determinant is near zero, ray lies in plane of triangle
			const float det = dot(edge1, pvec);

			if(det > -HIT_EPSILON && det < HIT_EPSILON)
				continue;

			const float inv_det = 1.0f / det;

			// calculate distance from vert0 to ray origin
			const vec3 tvec = ray.o - triv.v[0];

			// calculate U parameter and test bounds
			const float u = dot(tvec, pvec) * inv_det;
			if(u < 0.0f || u > 1.0f)
				continue;

			// prepare to test V parameter
			const vec3 qvec = cross(tvec, edge1);

			// calculate V parameter and test bounds
			const float v = dot(ray.d, qvec) * inv_det;
			if(v < 0.0f || u + v > 1.0f)
				continue;

			// calculate t, ray hits triangle
			const float f = dot(edge2, qvec) * inv_det;

			if(f >= g_rays.tmaxs[ray.id] || f < -HIT_EPSILON)
				continue;

			// store valid hit
			g_rays.tmaxs[ray.id] = f;
			g_rays.hits[ray.id] = t;
		}
	}
	++s_numIntersectCalls;
}

// ALTERNATIVES:
// https://github.com/hpicgs/cgsee/wiki/Ray-Box-Intersection-on-the-GPU
// http://citeseerx.ist.psu.edu/viewdoc/download?doi=10.1.1.147.2010&rep=rep1&type=pdf
// http://www.flipcode.com/archives/SSE_RayBox_Intersection_Test.shtml
// SIMD para varios raios ao mesmo tempo (precisa de SoA)
void filterRays(const Box& box, int& rend)
{
	int rendNew = 0;
	for(int r = 0; r < rend; ++r)
	{
		const Ray& ray = g_rays.rays[r];
		const vec3 invDir = reciprocal(ray.d);

		const float tx1 = (box.min.x - ray.o.x) * invDir.x;
		const float tx2 = (box.max.x - ray.o.x) * invDir.x;

		float tmin = std::min(tx1, tx2);
		float tmax = std::max(tx1, tx2);

		const float ty1 = (box.min.y - ray.o.y) * invDir.y;
		const float ty2 = (box.max.y - ray.o.y) * invDir.y;

		tmin = std::max(tmin, std::min(ty1, ty2));
		tmax = std::min(tmax, std::max(ty1, ty2));

		const float tz1 = (box.min.z - ray.o.z) * invDir.z;
		const float tz2 = (box.max.z - ray.o.z) * invDir.z;

		tmin = std::max(tmin, std::min(tz1, tz2));
		tmax = std::min(g_rays.tmaxs[ray.id], std::min(tmax, std::max(tz1, tz2)));

		if(tmin > tmax) continue;

		std::swap(g_rays.rays[r], g_rays.rays[rendNew]);
		++rendNew;
	}
	rend = rendNew;
}

void split(const Box& box, int& axis, float& pos)
{
	const float dx = box.max.x - box.min.x;
	const float dy = box.max.y - box.min.y;
	const float dz = box.max.z - box.min.z;

	axis = (dx > dy && dx > dz)? 0 : (dy > dz)? 1 : 2;
	pos = (box.min[axis] + box.max[axis]) * 0.5f;
}

void partitionTris(int tbegin, int tend, int axis, float pos, int& tsplit, Box& boxLeftTris, Box& boxLeftCenters, Box& boxRightTris, Box& boxRightCenters)
{
	tsplit = tbegin;
	for(int t = tbegin; t < tend; ++t)
	{
		const vec3& center = g_tris.centers[t];
		if(center[axis] < pos)
		{
			boxLeftTris.expand(g_tris.verts[t]);
			boxLeftCenters.expand(center);

			std::swap(g_tris.verts[t], g_tris.verts[tsplit]);
			std::swap(g_tris.centers[t], g_tris.centers[tsplit]);
			std::swap(g_tris.ids[t], g_tris.ids[tsplit]);
			++tsplit;
		}
		else
		{
			boxRightTris.expand(g_tris.verts[t]);
			boxRightCenters.expand(center);
		}
	}
}

void trace(const Box& boxTris, const Box& boxCenters, int tbegin, int tend, int rend)
{
	TIMED_CALL(filterRays(boxTris, rend), s_filterRayTime);

#ifdef HIT_CPU
	if((tend-tbegin) < 8 || rend < 8) // todo: define better
	{
		TIMED_CALL(intersectCPU(tbegin, tend, rend), s_intersectTime);
		return;
	}
#elif defined HIT_GPU
	if((tend-tbegin) < 8192 && rend < 32768)  // todo: define better
	{
		static timer __t;
		__t.restart();
		intersectGPU(tbegin, tend, rend);
		s_intersectTime += __t.msec();
		return;
	}
#endif

	int axis;
	float pos;
	TIMED_CALL(split(boxCenters, axis, pos), s_splitTime);

	Box boxLeftTris;
	Box boxLeftCenters;
	Box boxRightTris;
	Box boxRightCenters;
	int tsplit;
	TIMED_CALL(partitionTris(tbegin, tend, axis, pos, tsplit, boxLeftTris, boxLeftCenters, boxRightTris, boxRightCenters), s_partitionTriTime);

	if(g_rays.rays[0].d[axis] >= 0)
	{
		trace(boxLeftTris, boxLeftCenters, tbegin, tsplit, rend);
		trace(boxRightTris, boxRightCenters, tsplit, tend, rend);
	}
	else
	{
		trace(boxRightTris, boxRightCenters, tsplit, tend, rend);
		trace(boxLeftTris, boxLeftCenters, tbegin, tsplit, rend);
	}
}

// ---------------------------------------------------------------------------------------------------------------------------------------------------------------------------

struct Canvas
{
	int w;
	int h;
};

struct Camera
{
	vec3 position;
	vec3 lowerLeftDir;
	vec3 du;
	vec3 dv;
	int nu;
	int nv;
};

// ---------------------------------------------------------------------------------------------------------------------------------------------------------------------------

static Canvas g_canvas;
static Camera g_camera;

// ---------------------------------------------------------------------------------------------------------------------------------------------------------------------------

void generatePrimary()
{
	vec3 dir = g_camera.lowerLeftDir;
	int id = 0;

	for(int v = 0; v < g_camera.nv; ++v)
	{
		for(int u = 0; u < g_camera.nu; ++u)
		{
			Ray& r = g_rays.rays[id];
			r.o = g_camera.position;
			r.d = dir;
			r.id = id;
			dir += g_camera.du;
			++id;
		}
		dir += g_camera.dv;
	}

	for(int i = 0; i < g_rays.count; ++i)
	{
		g_rays.tmaxs[i] = FLT_MAX;
		g_rays.hits[i]  = -1;
	}

	hipMemcpy(d_tmaxs, g_rays.tmaxs, g_rays.count*sizeof(float), hipMemcpyDefault);
	hipMemcpy(d_hits, g_rays.hits, g_rays.count*sizeof(int), hipMemcpyDefault);
}

void shadePixels(unsigned char* pixels)
{
	hipMemcpy(g_rays.tmaxs, d_tmaxs, g_rays.count*sizeof(float), hipMemcpyDefault);
	hipMemcpy(g_rays.hits, d_hits, g_rays.count*sizeof(int), hipMemcpyDefault);

	for(int i = 0; i < g_rays.count; ++i)
	{
		pixels[i*3+0] = 0;
		pixels[i*3+1] = 0;
		pixels[i*3+2] = 0;
	}

	for(int i = 0; i < g_rays.count; ++i)
	{
		const Ray& ray = g_rays.rays[i];
		const float tmax = g_rays.tmaxs[ray.id];
		if(tmax < FLT_MAX)
		{
			// recompute hit position
			const vec3 hitPos = ray.o + ray.d * tmax;

			// recover triangle hit
			const int tid = g_rays.hits[ray.id];

			// recompute barycentric coordinates
			const TriV& tri = g_tris.verts[tid];
			const vec3 e0 = tri.v[1] - tri.v[0];
			const vec3 e1 = tri.v[2] - tri.v[0];
			const vec3 e2 = hitPos - tri.v[0];
			const float d00 = dot(e0, e0);
			const float d01 = dot(e0, e1);
			const float d11 = dot(e1, e1);
			const float d20 = dot(e2, e0);
			const float d21 = dot(e2, e1);
			const float invDenom = 1.0f / (d00 * d11 - d01 * d01);
			const float v = (d11 * d20 - d01 * d21) * invDenom;
			const float w = (d00 * d21 - d01 * d20) * invDenom;
			const float u = 1.0f - v - w;

			// lerp normal at hit position using barycentric coordinates
			const TriN& trin = g_tris.norms[g_tris.ids[tid]];
			const vec3 lerpN = normalize(trin.n[0]*u + trin.n[1]*v + trin.n[2]*w);

			// compute final color
			const unsigned char c = 255 * dot(lerpN, -normalize(ray.d));

			pixels[ray.id*3+0] = c;
			pixels[ray.id*3+1] = c;
			pixels[ray.id*3+2] = c;
		}
	}
}

// ---------------------------------------------------------------------------------------------------------------------------------------------------------------------------

void reshape(int w, int h)
{
	g_canvas.w = w;
	g_canvas.h = h;

	const int npixels = g_canvas.w * g_canvas.h;
	g_rays.count = npixels;

	// CPU --------------------------------------------------------

//	delete[] g_rays.ids;
//	g_rays.ids = new int[g_rays.count];
//	for(int i = 0; i < g_rays.count; ++i) g_rays.ids[i] = i;

	hipHostFree(g_rays.rays);
	hipHostMalloc(&g_rays.rays, g_rays.count*sizeof(Ray));

	hipHostFree(g_rays.tmaxs);
	hipHostMalloc(&g_rays.tmaxs, g_rays.count*sizeof(float));

	hipHostFree(g_rays.hits);
	hipHostMalloc(&g_rays.hits, g_rays.count*sizeof(int));

	// CPU pinned & GPU --------------------------------------------

	hipHostFree(h_rays);
	hipHostMalloc(&h_rays, g_rays.count*sizeof(Ray));

	hipFree(d_rays);
	hipMalloc(&d_rays, g_rays.count*sizeof(Ray));

	hipFree(d_tmaxs);
	hipMalloc(&d_tmaxs, g_rays.count*sizeof(float));

	hipFree(d_hits);
	hipMalloc(&d_hits, g_rays.count*sizeof(int));
}

void camera(float* peye, float* pcenter, float* pup)
{
	const vec3 eye = vec3(peye);
	const vec3 center = vec3(pcenter);
	const vec3 up = vec3(pup);

	// store position
	g_camera.position = eye;

	// pre-computations
	float invHeight = 1.0f / g_canvas.h;
	float invWidth  = 1.0f / g_canvas.w;

	// compute camera basis
	vec3 axisW = normalize(eye - center);
	vec3 axisV = normalize(up);
	vec3 axisU = cross(axisV, axisW);

	// compute half scale factors for each basis vector
	float sw = g_canvas.w * 0.01f; // try to keep directions around zero in floating-point value
	float sv = sw * std::tan(0.523598775f); // half 60o in radians
	float su = sv * g_canvas.w * invHeight;

	// scale each vector
	axisW *= sw;
	axisV *= sv;
	axisU *= su;

	// store final direction
	g_camera.lowerLeftDir = - axisU - axisV - axisW;

	// compute full scales
	axisV *= 2.0f;
	axisU *= 2.0f;

	// interpolation deltas
	g_camera.dv = axisV * invHeight - axisU; // also goes back to start of u-axis
	g_camera.du = axisU * invWidth;

	// number of pixels in U and V directions
	g_camera.nu = g_canvas.w;
	g_camera.nv = g_canvas.h;
}

void render(unsigned char* pixels)
{
	timer t;
	s_intersectTime = 0.0;
	s_triSendTime = 0.0;
	s_raySendTime = 0.0;
	s_kernelTime = 0.0;
	s_splitTime = 0.0;
	s_partitionTriTime = 0.0;
	s_filterRayTime = 0.0;
	s_numIntersectCalls = 0;

	std::cout << "---------------------------------" << std::endl;

	t.restart();
	generatePrimary();
	std::cout << "generate: " << std::setw(15) << (int)t.msec() << " ms" << std::endl;

	t.restart();
	trace(g_tris.boxTris, g_tris.boxCenters, 0, g_tris.count, g_rays.count);
	std::cout << "trace:    " << std::setw(15) << (int)t.msec() << " ms" << std::endl;

	std::cout << "  " << "filterRay: " << std::setw(9) << (int)s_filterRayTime << " ms" << std::endl;

	std::cout << "  " << "split:        " << std::setw(9) << (int)s_splitTime << " ms" << std::endl;
	std::cout << "  " << "intersect:    " << std::setw(9) << (int)s_intersectTime << " ms (count: " << s_numIntersectCalls << ", avg: " << s_intersectTime / s_numIntersectCalls << " ms)" << std::endl;

	std::cout << "    " << "triSend:    " << std::setw(9) << s_triSendTime << " ms" << std::endl;
	std::cout << "    " << "raySend:    " << std::setw(9) << s_raySendTime << " ms" << std::endl;
	std::cout << "    " << "kernel:    " << std::setw(9) << s_kernelTime << " ms (avg: " << s_kernelTime / s_numIntersectCalls << " ms)" << std::endl;

	std::cout << "  " << "partitionTri: " << std::setw(9) << (int)s_partitionTriTime << " ms" << std::endl;

	t.restart();
	shadePixels(pixels);
	std::cout << "shade:    " << std::setw(15) << (int)t.msec() << " ms" << std::endl;
}

//---------------------------------------------------------------------------------------------------------------------

void loadScene1tri()
{
	TriV triv;
	triv.v[0] = vec3(-1,-1,0);
	triv.v[1] = vec3(1,-1,0);
	triv.v[2] = vec3(0,1,0);
	hipHostMalloc(&g_tris.verts, 1*sizeof(TriV));
	g_tris.verts[0] = triv;

	TriN trin;
	trin.n[0] = normalize(cross(triv.v[1] - triv.v[0], triv.v[2]-triv.v[0]));
	trin.n[1] = trin.n[0];
	trin.n[2] = trin.n[0];
	g_tris.norms = new TriN[1];
	g_tris.norms[0] = trin;

	g_tris.count = 1;
}

float randf(float min, float max)
{
	return min + (float)rand()/(float)RAND_MAX * (max - min);
}

void loadSceneRand()
{
	g_tris.count = 100000;
	hipHostMalloc(&g_tris.verts, g_tris.count*sizeof(TriV));
	g_tris.norms = new TriN[g_tris.count];
	int dst = 0;

	srand(122);

	for(int i = 0; i < g_tris.count; ++i)
	{
		vec3 c(randf(-5.0f, 5.0f), randf(-5.0f, 5.0f), randf(-5.0f, 5.0f));
		TriV triv;
		triv.v[0] = c + vec3(randf(0.1f, 0.5f), randf(0.1f, 0.5f), randf(0.1f, 0.5f));
		triv.v[1] = c + vec3(randf(0.1f, 0.5f), randf(0.1f, 0.5f), randf(0.1f, 0.5f));
		triv.v[2] = c + vec3(randf(0.1f, 0.5f), randf(0.1f, 0.5f), randf(0.1f, 0.5f));
		g_tris.verts[dst] = triv;

		TriN trin;
		trin.n[0] = normalize(cross(triv.v[1] - triv.v[0], triv.v[2]-triv.v[0]));
		trin.n[1] = trin.n[0];
		trin.n[2] = trin.n[0];
		g_tris.norms[dst] = trin;

		++dst;
	}
}

static std::vector<vec3> vertices;
static std::vector<int> elements;

static int vertex_cb(p_ply_argument argument)
{
	long id;
	ply_get_argument_user_data(argument, NULL, &id);
	if(id == 0)
	{
		vertices.resize(vertices.size()+1);
	}
	vertices.back()[id] = ply_get_argument_value(argument) * 50;
	return 1;
}

static int face_cb(p_ply_argument argument)
{
	long length, value_index;
	ply_get_argument_property(argument, NULL, &length, &value_index);
	switch (value_index)
	{
	case 0:
	case 1:
	case 2:
		elements.push_back(ply_get_argument_value(argument));
		break;
	default:
		break;
	}
	return 1;
}

void loadSceneBunny()
{
	p_ply ply = ply_open("/home/environ/Downloads/bunny.ply", NULL, 0, NULL);
	if (!ply) exit(1);
	if (!ply_read_header(ply)) exit(1);
	long nvertices = ply_set_read_cb(ply, "vertex", "x", vertex_cb, NULL, 0);
	ply_set_read_cb(ply, "vertex", "y", vertex_cb, NULL, 1);
	ply_set_read_cb(ply, "vertex", "z", vertex_cb, NULL, 2);
	long ntriangles = ply_set_read_cb(ply, "face", "vertex_indices", face_cb, NULL, 0);
	if (!ply_read(ply)) exit(1);
	ply_close(ply);

	g_tris.count = elements.size()/3;
	hipHostMalloc(&g_tris.verts, g_tris.count*sizeof(TriV));
	g_tris.norms = new TriN[g_tris.count];

	std::vector<vec3> normals(vertices.size());

	for(unsigned int e = 0; e < elements.size(); e+=3)
	{
		const int e0 = elements[e+0];
		const int e1 = elements[e+1];
		const int e2 = elements[e+2];

		vec3 v0 = vertices[e0];
		vec3 v1 = vertices[e1];
		vec3 v2 = vertices[e2];

		vec3 n = cross(v1-v0, v2-v0);

		normals[e0] += n;
		normals[e1] += n;
		normals[e2] += n;
	}

	int dst = 0;
	for(unsigned int e = 0; e < elements.size(); e+=3)
	{
		const int e0 = elements[e+0];
		const int e1 = elements[e+1];
		const int e2 = elements[e+2];

		TriV triv;
		triv.v[0] = vertices[e0];
		triv.v[1] = vertices[e1];
		triv.v[2] = vertices[e2];
		g_tris.verts[dst] = triv;

		TriN trin;
		trin.n[0] = normalize(normals[e0]);
		trin.n[1] = normalize(normals[e1]);
		trin.n[2] = normalize(normals[e2]);
		g_tris.norms[dst] = trin;

		++dst;
	}
}

void endLoadScene()
{
	g_tris.ids = new int[g_tris.count];
	g_tris.boxes = new Box[g_tris.count];
	g_tris.centers = new vec3[g_tris.count];
	for(int i = 0; i < g_tris.count; ++i)
	{
		g_tris.ids[i] = i;
		const TriV& triv = g_tris.verts[i];
		g_tris.boxes[i].expand(triv);
		g_tris.centers[i] = (triv.v[0] + triv.v[1] + triv.v[2]) * 0.333333333333f;
		g_tris.boxTris.expand(triv);
		g_tris.boxCenters.expand(g_tris.centers[i]);
	}

	hipHostMalloc(&h_tris, g_tris.count*sizeof(TriV));
	hipMalloc(&d_tris, g_tris.count*sizeof(TriV));

	vec3 center = (g_tris.boxTris.min + g_tris.boxTris.max) * 0.5f;
	vec3 eye = center + vec3(0,0,10);
	vec3 up(0,1,0);
	rtvSetCamera(eye.ptr(), center.ptr(), up.ptr());
}

//---------------------------------------------------------------------------------------------------------------------

int main()
{
	rtvInit(1024, 1024);
	rtvSetReshapeCallback(reshape);
	rtvSetCameraCallback(camera);
	rtvSetRenderCallback(render);

//	loadScene1tri();
//	loadSceneRand();
	loadSceneBunny();

	endLoadScene();

	rtvExec();

	return 0;
}
